#include "hip/hip_runtime.h"
// MODIFIED INMA
#include "stencil.h"
#include "amul.h"

extern "C" __global__ void
mdatatemp(float* __restrict__ dst_sinx, float* __restrict__ dst_siny, float* __restrict__ dst_sinz,
          float* __restrict__ dst_cosx, float* __restrict__ dst_cosy, float* __restrict__ dst_cosz,
          float* __restrict__ delta_time, float* __restrict__ brms_x, float* __restrict__ brms_y, float* __restrict__ brms_z,
          float* __restrict__ current_mx, float* __restrict__ current_my, float* __restrict__ current_mz,
          float ctimeWc, float h_delta, float brmsx, float brmsy, float brmsz, int N) {

          int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

          if (i < N) {

              // First Summatory
              dst_sinx[i] += amul(current_mx, sin(ctimeWc), i);
              dst_siny[i] += amul(current_my, sin(ctimeWc), i);
              dst_sinz[i] += amul(current_mz, sin(ctimeWc), i);

              dst_cosx[i] += amul(current_mx, cos(ctimeWc), i);
              dst_cosy[i] += amul(current_my, cos(ctimeWc), i);
              dst_cosz[i] += amul(current_mz, cos(ctimeWc), i);

              delta_time[i] = h_delta;
              brms_x[i] = brmsx;
              brms_y[i] = brmsy;
              brms_z[i] = brmsz;
          }
}
