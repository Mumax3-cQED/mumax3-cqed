
#include <hip/hip_runtime.h>

extern "C" __global__ void
mdatatemp(float* __restrict__  dst_x, float* __restrict__  dst_y, float* __restrict__  dst_z, float* __restrict__ sin_full_time,
      float* __restrict__  mx_temp, float* __restrict__  my_temp, float* __restrict__  mz_temp, float wc, float full_tau, float dt, int N) {

    	// for (int i = 0; i < size_x*size_y*size_z; i++) {
      int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

      if(i < N) {

    				dst_x[i] += mx_temp[i];
    				dst_y[i] += my_temp[i];
    				dst_z[i] += mz_temp[i];
            sin_full_time[i] = sin(wc * (full_tau - dt));
      }
}
