#include "hip/hip_runtime.h"
// MODIFIED INMA

#include "amul.h"
#include "float3.h"
#include <stdint.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

__device__ __constant__ double HBAR = 1.054571817E-34;

static __inline__ __device__ float3 operator*(const float3 &a, const float3 &b) {
  return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

// Landau-Lifshitz torque.
//- 1/(1+α²) [ m x B +  α m x (m x B) ]
extern "C" __global__ void
lltorque2time(float* __restrict__  tx, float* __restrict__  ty, float* __restrict__  tz,
          float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
          float* __restrict__  hx, float* __restrict__  hy, float* __restrict__  hz,
          float* __restrict__  alpha_, float alpha_mul,
          float delta_time, float wc, float brms_x, float brms_y, float brms_z,
          float* __restrict__ brmsi_x, float* __restrict__ brmsi_y, float* __restrict__ brmsi_z,
          float* __restrict__ rk_sin_mx, float* __restrict__ rk_sin_my, float* __restrict__ rk_sin_mz,
          float* __restrict__ rk_cos_mx, float* __restrict__ rk_cos_my, float* __restrict__ rk_cos_mz, float* __restrict__ ctime, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if (i < N) {

        float3 m = {mx[i], my[i], mz[i]};
        float3 H = {hx[i], hy[i], hz[i]};

        float alpha = amul(alpha_, alpha_mul, i);

        float3 mxH = cross(m, H);
        float gilb = -1.0f / (1.0f + alpha * alpha);
        // float3 torque = gilb * (mxH + alpha * cross(m, mxH)); // LLG equation

        // Adding new time-dependant term to equations
        brmsi_x[i] = brms_x;
        brmsi_y[i] = brms_y;
        brmsi_z[i] = brms_z;

        float3 brms = {brmsi_x[i] , brmsi_y[i], brmsi_z[i]};

        float3 mxBrms = cross(m, brms); // m x Brms

        float3 rk_sin_m = {rk_sin_mx[i], rk_sin_my[i], rk_sin_mz[i]};
        float3 rk_cos_m = {rk_cos_mx[i], rk_cos_my[i], rk_cos_mz[i]};

        // Intergal from 0 to t
        float3 si_sum_total = delta_time * ((cos(wc * ctime[i]) * rk_sin_m) - (sin(wc * ctime[i]) * rk_cos_m));

        // Summatory for all cells
        float3 sum_final = brms * si_sum_total;

        float hbar_const = (2 / HBAR);
        float3 new_term = (hbar_const * mxBrms * sum_final);

        float3 torque = (gilb * (mxH + alpha * cross(m, mxH))) - new_term; // LLG equation with full new time-dependant term to plug in equation

        // float3 torque = gilb * (mxH + alpha * cross(m, mxH)); // LLG equation

        tx[i] = torque.x;
        ty[i] = torque.y;
        tz[i] = torque.z;
    }
}
