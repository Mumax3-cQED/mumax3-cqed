#include "hip/hip_runtime.h"
// MODIFIED INMA

#include "amul.h"
#include "float3.h"
#include <stdint.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
#include "constants.h"

#define GS 2.0

static __inline__ __device__ float3 operator*(const float3 &a, const float3 &b) {
  return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

// Landau-Lifshitz torque.
//- 1/(1+α²) [ m x B +  α m x (m x B) ]
extern "C" __global__ void
lltorque2time(float* __restrict__  tx, float* __restrict__  ty, float* __restrict__  tz,
          float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
          float* __restrict__  hx, float* __restrict__  hy, float* __restrict__  hz,
          float* __restrict__  alpha_, float alpha_mul,
          float delta_time, float wc, float brms_x, float brms_y, float brms_z,
          float* __restrict__ brmsi_x, float* __restrict__ brmsi_y, float* __restrict__ brmsi_z,
          float* __restrict__ rk_sin_mx, float* __restrict__ rk_sin_my, float* __restrict__ rk_sin_mz,
          float* __restrict__ rk_cos_mx, float* __restrict__ rk_cos_my, float* __restrict__ rk_cos_mz, float* __restrict__ ctime,
          float* __restrict__ sum_cell_x, float* __restrict__ sum_cell_y, float* __restrict__ sum_cell_z, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if (i < N) {

        float3 m = {mx[i], my[i], mz[i]};
        float3 H = {hx[i], hy[i], hz[i]};
        float alpha = amul(alpha_, alpha_mul, i);

        float3 mxH = cross(m, H);
        float gilb = -1.0f / (1.0f + alpha * alpha);
        // float3 torque = gilb * (mxH + alpha * cross(m, mxH)); // LLG equation

        // Adding new time-dependant term to equations
        brmsi_x[i] = brms_x;
        brmsi_y[i] = brms_y;
        brmsi_z[i] = brms_z;

        float3 brms = {brmsi_x[i] , brmsi_y[i], brmsi_z[i]};

        float3 mxBrms = cross(m, brms); // m x Brms

        float3 rk_sin_m = {rk_sin_mx[i], rk_sin_my[i], rk_sin_mz[i]};
        float3 rk_cos_m = {rk_cos_mx[i], rk_cos_my[i], rk_cos_mz[i]};

        // Summatory for all cells
        float3 si_sum_total = delta_time * ((cos(wc * ctime[i]) * rk_sin_m) - (sin(wc * ctime[i]) * rk_cos_m));

        sum_cell_x[i] += brms.x * si_sum_total.x;
        sum_cell_y[i] += brms.y * si_sum_total.y;
        sum_cell_z[i] += brms.z * si_sum_total.z;

        float3 sum_final = {sum_cell_x[i], sum_cell_y[i], sum_cell_z[i]};

        float spin_constant = 2 / HBAR; // debemos dividir entre gamma0 nuestro nuevo termino? parece que si
        float3 new_term = (spin_constant * mxBrms * sum_final);

        float3 torque = gilb * (mxH + alpha * cross(m, mxH)) - new_term;  // LLG equation with full new time-dependant term to plug in equation

        tx[i] = torque.x;
        ty[i] = torque.y;
        tz[i] = torque.z;
    }
}
