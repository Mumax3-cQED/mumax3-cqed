#include "hip/hip_runtime.h"
// MODIFIED INMA
#include "amul.h"
#include "constants.h"
#include "stencil.h"
#include <stdint.h>

extern "C" __global__ void
addspin2beff(float* __restrict__ tx, float* __restrict__ ty, float* __restrict__ tz,
          float* __restrict__ dst_sin_x,
          float* __restrict__ dst_sin_y,
          float* __restrict__ dst_sin_z,
          float* __restrict__ dst_cos_x,
          float* __restrict__ dst_cos_y,
          float* __restrict__ dst_cos_z,
          float* __restrict__ wc, float wc_mul, float msat,
          float* __restrict__ brms_x, float brmsx_mul,
          float* __restrict__ brms_y, float brmsy_mul,
          float* __restrict__ brms_z, float brmsz_mul,
          float* __restrict__ mx,
          float* __restrict__ my,
          float* __restrict__ mz,
          float delta_time, float ctime, float vol, int N) {

        int i = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

        if (i < N) {

          float wc_val = amul(wc, wc_mul, i);

          float brmsx = amul(brms_x, brmsx_mul, i);
          float brmsy = amul(brms_y, brmsy_mul, i);
          float brmsz = amul(brms_z, brmsz_mul, i);

          // First summatory
          dst_sin_x[i] += amul(mx, sin(ctime * wc_val), i);
          dst_sin_y[i] += amul(my, sin(ctime * wc_val), i);
          dst_sin_z[i] += amul(mz, sin(ctime * wc_val), i);

          dst_cos_x[i] += amul(mx, cos(ctime * wc_val), i);
          dst_cos_y[i] += amul(my, cos(ctime * wc_val), i);
          dst_cos_z[i] += amul(mz, cos(ctime * wc_val), i);

          __syncthreads();

          // Second summatory
          float result_sum = 0.0;

          for (int ii = (blockIdx.y * blockDim.y + threadIdx.y) * blockDim.x * gridDim.x + (blockIdx.x * blockDim.x + threadIdx.x);
              ii < N;
              ii += blockDim.y * gridDim.y * blockDim.x * gridDim.x) {

            float sum_x = brmsx * (delta_time/GAMMA0) * ((dst_sin_x[ii] * cos(ctime * wc_val) - dst_cos_x[ii] * sin(ctime * wc_val)));
            float sum_y = brmsy * (delta_time/GAMMA0) * ((dst_sin_y[ii] * cos(ctime * wc_val) - dst_cos_y[ii] * sin(ctime * wc_val)));
            float sum_z = brmsz * (delta_time/GAMMA0) * ((dst_sin_z[ii] * cos(ctime * wc_val) - dst_cos_z[ii] * sin(ctime * wc_val)));

            result_sum += (sum_x + sum_y + sum_z);
          }

          __syncthreads();

          float prefactor = (2 / HBAR) * vol * msat;
          float3 brms = {brmsx, brmsy, brmsz};

          float3 torque = prefactor * result_sum * brms;

          //printf("torque.x: %.8f\n", torque.x);
          //printf(torque.y: "%.8f\n", torque.y);
          //printf(torque.z: "%.8f\n", torque.z);

          // Final value
          tx[i] -= torque.x;
          ty[i] -= torque.y;
          tz[i] -= torque.z;
       }
}
