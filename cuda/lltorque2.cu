#include "hip/hip_runtime.h"
#include "amul.h"
#include "float3.h"
#include <stdint.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

__device__ __constant__ double MUB = 9.2740091523E-24;
__device__ __constant__ double HBAR = 1.054571817E-34;
__device__ __constant__ double GS = 2.0;

//#define CONSTANT (powf(GS,2)*powf(MUB,2))/(powf(HBAR,3))

 // __device__ float d_si_sum_total = 0.0;
// __device__ int exec_threads = 0;


// inline __device__ __device__ float3 operator--(float3 a, float3 b)
// {
//     return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
// }

__device__ float spin_torque(float wc_frec, float mx_val, float my_val, float mz_val) {

  float sum_term = mx_val * wc_frec + my_val * wc_frec + mz_val * wc_frec;
  return sum_term;
}

// Landau-Lifshitz torque.
//- 1/(1+α²) [ m x B +  α m x (m x B) ]
extern "C" __global__ void
lltorque2(float* __restrict__  tx, float* __restrict__  ty, float* __restrict__  tz,
          float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
          float* __restrict__  hx, float* __restrict__  hy, float* __restrict__  hz,
          float* __restrict__  alpha_, float alpha_mul, int N, float dt, float fixed_dt, float time, float wc, float brms_x, float brms_y, float brms_z, float* __restrict__ deltas) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if (i < N) {

        int idx = i;
        deltas[i] = dt;

        float3 m = {mx[i], my[i], mz[i]};
        float3 H = {hx[i], hy[i], hz[i]};

        float alpha = amul(alpha_, alpha_mul, i);

        float3 mxH = cross(m, H);
        float gilb = -1.0f / (1.0f + alpha * alpha);
        // float3 torque = gilb * (mxH + alpha * cross(m, mxH));

        //double h_bar = 1.054571817E-34; // h-bar planck value
        //double muB = 9.274009994E-24; // Bohr magneton
        //double gs = 2.0;
        float constant_term = (float)(powf(GS,2)*powf(MUB,2))/(powf(HBAR,3)); //  2.9334e+56;
        //float constant_term = (float)powf(gs,2)*pow(muB,2);//(float)(powf(gs,2)*powf(muB,2))/powf(h_bar,3);
      //  constant_term = fdividef(constant_term, powf(h_bar, 3));

        float3 brms = {brms_x , brms_y, brms_z};
        float3 mxBrms = cross(m, brms); // Si = m

        float si_sum_total = 0.0;

        for (int z = 0; z <= idx; z++) {
      //  for (float dtz = 0.0; dtz <= time; dtz+=dt) {
          si_sum_total += spin_torque(sin(wc*(time - deltas[z])), mx[z], my[z], mz[z]) * fixed_dt;
        }
        // si_sum_total = d_si_sum_total;

        // float value_sum = d_si_sum_total;
        //
        // d_si_sum_total = si_sum_total;
        // d_si_sum_total += (si_sum_total + val_sim_sum_total); //????
        //d_si_sum_total += val_sim_sum_total;
        //si_sum_total = d_si_sum_total;

        float full_term_zero = 0.0;
        float full_term_one = 0.0;
        float full_term_two = 0.0;

        for (int z = 0; z <= idx; z++) {
          full_term_zero += brms.x * si_sum_total;
          full_term_one +=  brms.y * si_sum_total;
          full_term_two +=  brms.z * si_sum_total;
        }

        // float3 items_term = {full_term_zero, full_term_one, full_term_two};
        float vect_modulus = sqrt(pow(full_term_zero, 2) + pow(full_term_one, 2) + pow(full_term_two, 2));

        float3 append_term = 2 * mxBrms * vect_modulus;
        // append_term = append_term * constant_term;

        float3 torque = (gilb * (mxH + alpha * cross(m, mxH))) - (append_term);

    //float3 torque = gilb * (mxH + alpha * cross(m, mxH));
        tx[i] = torque.x;
        ty[i] = torque.y;
        tz[i] = torque.z;

    //    __syncthreads();
    }
}
