#include "hip/hip_runtime.h"
#include "amul.h"
#include "float3.h"
#include <stdint.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

__device__ __constant__ double MUB = 9.2740091523E-24;
__device__ __constant__ double HBAR = 1.054571817E-34;
__device__ __constant__ double GS = 2.0;

__device__ float spinTorque(float calc_term, float mx_val, float my_val, float mz_val) {

  float sum_term = mx_val * calc_term + my_val * calc_term + mz_val * calc_term;
  return sum_term;
}

// Landau-Lifshitz torque.
//- 1/(1+α²) [ m x B +  α m x (m x B) ]
extern "C" __global__ void
lltorque2(float* __restrict__  tx, float* __restrict__  ty, float* __restrict__  tz,
          float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
          float* __restrict__  hx, float* __restrict__  hy, float* __restrict__  hz,
          float* __restrict__  alpha_, float alpha_mul, int N, float dt, float fixed_dt, float time, float wc, float brms_x, float brms_y, float brms_z, float* __restrict__ deltas) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if (i < N) {

        int idx = i;

        float3 m = {mx[i], my[i], mz[i]};
        float3 H = {hx[i], hy[i], hz[i]};

        float alpha = amul(alpha_, alpha_mul, i);

        float3 mxH = cross(m, H);
        float gilb = -1.0f / (1.0f + alpha * alpha);

        float3 brms = {brms_x , brms_y, brms_z};
        float3 mxBrms = cross(m, brms); // Si = m

        deltas[i] = dt;

        __syncthreads();

        float si_sum_total = 0.0;

        for (int z = 0; z <= idx; z++) {

          float single_delta = deltas[z];

          if (single_delta > 0) {
              si_sum_total += spinTorque(sin(wc*(time - single_delta)), mx[z], my[z], mz[z]) * fixed_dt;
          }
        }

        float ivect = 0.0;
        float jvect = 0.0;
        float kvect = 0.0;

        for (int z = 0; z <= idx; z++) {
          ivect += (brms.x * si_sum_total);
          jvect += (brms.y * si_sum_total);
          kvect += (brms.z * si_sum_total);
        }

        // float3 items_term = {full_term_zero, full_term_one, full_term_two};
        float vect_modulus = sqrt(pow(ivect, 2) + pow(jvect, 2) + pow(kvect, 2));

        float constant_term = (float)(pow(GS,2)*pow(MUB,2))/(pow(HBAR,3));

        float3 new_term = 2 * constant_term * mxBrms * vect_modulus;

        float3 torque = (gilb * (mxH + alpha * cross(m, mxH))) - (new_term);

        // float3 torque = gilb * (mxH + alpha * cross(m, mxH));

        tx[i] = torque.x;
        ty[i] = torque.y;
        tz[i] = torque.z;
    }
}
