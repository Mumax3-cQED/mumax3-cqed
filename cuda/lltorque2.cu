#include "hip/hip_runtime.h"
#include "amul.h"
#include "float3.h"
#include <stdint.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

__device__ __constant__ double MUB = 9.2740091523E-24;
__device__ __constant__ double HBAR = 1.054571817E-34;
__device__ __constant__ double GS = 2.0;

//#define CONSTANT (powf(GS,2)*powf(MUB,2))/(powf(HBAR,3))

 // __device__ float d_si_sum_total = 0.0;
// __device__ int exec_threads = 0;


// inline __device__ __device__ float3 operator--(float3 a, float3 b)
// {
//     return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
// }

__device__ float spin_torque(float calc_term, float mx_val, float my_val, float mz_val) {

  float sum_term = mx_val * calc_term + my_val * calc_term + mz_val * calc_term;
  return sum_term;
}

// Landau-Lifshitz torque.
//- 1/(1+α²) [ m x B +  α m x (m x B) ]
extern "C" __global__ void
lltorque2(float* __restrict__  tx, float* __restrict__  ty, float* __restrict__  tz,
          float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
          float* __restrict__  hx, float* __restrict__  hy, float* __restrict__  hz,
          float* __restrict__  alpha_, float alpha_mul, int N, float dt, float fixed_dt, float time, float wc, float brms_x, float brms_y, float brms_z, float* __restrict__ deltas) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if (i < N) {

        int idx = i;

        float3 m = {mx[i], my[i], mz[i]};
        float3 H = {hx[i], hy[i], hz[i]};

        float alpha = amul(alpha_, alpha_mul, i);

        float3 mxH = cross(m, H);
        float gilb = -1.0f / (1.0f + alpha * alpha);
        // float3 torque = gilb * (mxH + alpha * cross(m, mxH));

        float3 brms = {brms_x , brms_y, brms_z};
        float3 mxBrms = cross(m, brms); // Si = m

        deltas[i] = dt;

        __syncthreads();

        float si_sum_total = 0.0;

        for (int z = 0; z <= idx; z++) {

          float single_delta = deltas[z];

          if (single_delta > 0) {
              si_sum_total += spin_torque(sin(wc*(time - single_delta)), mx[z], my[z], mz[z]) * fixed_dt;
          }
        }

        float full_term_zero = 0.0;
        float full_term_one = 0.0;
        float full_term_two = 0.0;

        for (int z = 0; z <= idx; z++) {
          full_term_zero += brms.x * si_sum_total;
          full_term_one +=  brms.y * si_sum_total;
          full_term_two +=  brms.z * si_sum_total;
        }

        // float3 items_term = {full_term_zero, full_term_one, full_term_two};
        float vect_modulus = sqrt(pow(full_term_zero, 2) + pow(full_term_one, 2) + pow(full_term_two, 2));

        float constant_term = (float)(powf(GS,2)*powf(MUB,2))/(powf(HBAR,3)); //  2.9334e+56;

        float3 append_term = 2 * mxBrms * vect_modulus;
        // append_term = append_term * constant_term;

        float3 torque = (gilb * (mxH + alpha * cross(m, mxH))) - (append_term);

        // float3 torque = gilb * (mxH + alpha * cross(m, mxH));
        tx[i] = torque.x;
        ty[i] = torque.y;
        tz[i] = torque.z;

        // __syncthreads();
    }
}
