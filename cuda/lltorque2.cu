#include "hip/hip_runtime.h"
#include "amul.h"
#include "float3.h"
#include <stdint.h>
#include <iostream>
#include <stdio.h>
#include <math.h>

__device__ __constant__ double MUB = 9.2740091523E-24;
__device__ __constant__ double HBAR = 1.054571817E-34;
__device__ __constant__ double GS = 2.0;

// Landau-Lifshitz torque.
//- 1/(1+α²) [ m x B +  α m x (m x B) ]
extern "C" __global__ void
lltorque2(float* __restrict__  tx, float* __restrict__  ty, float* __restrict__  tz,
          float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
          float* __restrict__  hx, float* __restrict__  hy, float* __restrict__  hz,
          float* __restrict__  alpha_, float alpha_mul,
          float time, float fixed_dt, float wc, float brms_x, float brms_y, float brms_z,
          float* __restrict__ rk_mx, float* __restrict__ rk_my, float* __restrict__ rk_mz, float* __restrict__ rk_tau, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if (i < N) {

        float3 m = {mx[i], my[i], mz[i]};
        float3 H = {hx[i], hy[i], hz[i]};

        float alpha = amul(alpha_, alpha_mul, i);

        float3 mxH = cross(m, H);
        float gilb = -1.0f / (1.0f + alpha * alpha);
        // float3 torque = gilb * (mxH + alpha * cross(m, mxH)); // LLG equation

        // Adding new time-dependant term to equations
        float3 brms = {brms_x , brms_y, brms_z};
        float3 mxBrms = cross(m, brms); // m x Brms

        float3 rk_m = {rk_mx[i], rk_my[i], rk_mz[i]};

        // Intergal from 0 to t
        float3 si_sum_total = sin(wc*(time - rk_tau[i])) * rk_m * fixed_dt;

        // Summatory for all cells
        // https://developer.download.nvidia.com/cg/dot.html
        float sum_final = 0.0;

        for (int cell = 0; cell < N; cell++) {
          sum_final += dot(si_sum_total, brms);
        }

        float constant_term = 1; //(float)(pow(GS,2)*pow(MUB,2))/(pow(HBAR,3)); // Constant value (gs^2*mub^2)/hbar^3

        float3 new_term = 2 * constant_term * mxBrms * sum_final; // LLG equation with full new time-dependant term to plug in equation

        float3 torque = (gilb * (mxH + alpha * cross(m, mxH))) - (new_term);

        // float3 torque = gilb * (mxH + alpha * cross(m, mxH)); // LLG equation

        tx[i] = torque.x;
        ty[i] = torque.y;
        tz[i] = torque.z;
    }
}
