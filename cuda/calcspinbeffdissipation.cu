#include "hip/hip_runtime.h"
// CREATED AND MODIFIED INMA
#include "amul.h"
#include "float3.h"
#include <stdint.h>
#include "stencil.h"

// Calculations for extra term in Beff with cavity dissipation
extern "C" __global__ void
calcspinbeffdissipation(float* __restrict__  tx, float* __restrict__  ty, float* __restrict__  tz,
            float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
            float* __restrict__ snx,   float* __restrict__ sny, float* __restrict__ snz,
            float* __restrict__ cnx, float* __restrict__ cny, float* __restrict__ cnz,
            float* __restrict__ wc, float wc_mul,
            float* __restrict__ kappa, float kappa_mul,
            float* __restrict__ brms_x, float brmsx_mul,
            float* __restrict__ brms_y, float brmsy_mul,
            float* __restrict__ brms_z, float brmsz_mul,
            float nspins, float dt, float ctime, float gammaLL, int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
       return;
    }

    int i = idx(ix, iy, iz);

    float wc_val = amul(wc, wc_mul, i);

    float kappa_val = amul(kappa, kappa_mul, i);

    float brmsx = amul(brms_x, brmsx_mul, i);
    float brmsy = amul(brms_y, brmsy_mul, i);
    float brmsz = amul(brms_z, brmsz_mul, i);

    // Summatory
    float3 mi = make_float3(mx[i], my[i], mz[i]);
    float3 brmsi = make_float3(brmsx, brmsy, brmsz);

    snx[i] += exp(kappa_val * ctime) * sin(wc_val * ctime) * dot(mi, brmsi) * dt;
    cnx[i] += exp(kappa_val * ctime) * cos(wc_val * ctime) * dot(mi, brmsi) * dt;

    // snx[i] += exp(kappa_val * ctime) * sin(wc_val * ctime) * amul(mx, brmsx, i) * dt;
    // sny[i] += exp(kappa_val * ctime) * sin(wc_val * ctime) * amul(my, brmsy, i) * dt;
    // snz[i] += exp(kappa_val * ctime) * sin(wc_val * ctime) * amul(mz, brmsz, i) * dt;

    // cnx[i] += exp(kappa_val * ctime) * cos(wc_val * ctime) * amul(mx, brmsx, i) * dt;
    // cny[i] += exp(kappa_val * ctime) * cos(wc_val * ctime) * amul(my, brmsy, i) * dt;
    // cnz[i] += exp(kappa_val * ctime) * cos(wc_val * ctime) * amul(mz, brmsz, i) * dt;

    // Summatory
    // float sn = snx[i]; //+ sny[i] + snz[i];
    // float cn = cnx[i]; //+ cny[i] + cnz[i];

    float PREFACTOR = gammaLL * nspins;
    float G = PREFACTOR * exp(-kappa_val * ctime) * (cos(wc_val * ctime) * snx[i] - sin(wc_val * ctime) * cnx[i]);

    // This is the new term to Beff
    float new_term_x = brmsx * G;
    float new_term_y = brmsy * G;
    float new_term_z = brmsz * G;

    // Beff = Beff - new_term
    tx[i] -= new_term_x;
    ty[i] -= new_term_y;
    tz[i] -= new_term_z;
}
