#include "hip/hip_runtime.h"
// CREATED AND MODIFIED INMA
#include "amul.h"
#include "float3.h"
#include <stdint.h>
#include "stencil.h"

// Calculations for extra term in Beff without cavity dissipation
extern "C" __global__ void
calcspinbeff(float* __restrict__  tx, float* __restrict__  ty, float* __restrict__  tz,
            float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz,
            float* __restrict__ sn, float* __restrict__ cn, 
            float* __restrict__ wc, float wc_mul,
            float* __restrict__ brms_x, float brmsx_mul,
            float* __restrict__ brms_y, float brmsy_mul,
            float* __restrict__ brms_z, float brmsz_mul,
            float nspins, float dt, float ctime, float gammaLL, int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
       return;
    }

    int i = idx(ix, iy, iz);

    float wc_val = amul(wc, wc_mul, i);

    float brmsx = amul(brms_x, brmsx_mul, i);
    float brmsy = amul(brms_y, brmsy_mul, i);
    float brmsz = amul(brms_z, brmsz_mul, i);

    // Summatory
    float3 mi = make_float3(mx[i], my[i], mz[i]);
    float3 brmsi = make_float3(brmsx, brmsy, brmsz);

    sn[i] += sin(wc_val * ctime) * dot(mi, brmsi) * dt;
    cn[i] += cos(wc_val * ctime) * dot(mi, brmsi) * dt;

    float PREFACTOR = gammaLL * nspins;
    float G = PREFACTOR * (cos(wc_val * ctime) * sn[i] - sin(wc_val * ctime) * cn[i]);

    // This is the new term to Beff
    float new_term_x = brmsx * G;
    float new_term_y = brmsy * G;
    float new_term_z = brmsz * G;

    // Beff = Beff - new_term
    tx[i] -= new_term_x;
    ty[i] -= new_term_y;
    tz[i] -= new_term_z;
}
